#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include<opencv2/opencv.hpp>
#include <stdio.h>
#include <iostream>

using namespace std;
using namespace cv;

//��������
#define CHECK_ERROR(call){\
    const hipError_t err = call;\
    if (err != hipSuccess)\
    {\
        printf("Error:%s,%d,",__FILE__,__LINE__);\
        printf("code:%d,reason:%s\n",err,hipGetErrorString(err));\
        exit(1);\
    }\
}

//�ں˺�����ʵ�����·�ת
__global__ void swap_image_kernel(cuda::PtrStepSz<uchar3> cu_src, cuda::PtrStepSz<uchar3> cu_dst, int h, int w)
{
	//����ķ������ο�ǰ������
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
	//ΪɶҪ�������ƣ��ο�ǰ������
	if (x < cu_src.cols && y < cu_src.rows)
	{
		//Ϊ�β���h-y-1,������h-y���Լ�˼��Ŷ
		cu_dst(y, x) = cu_src(h - y - 1, x);
	}
}
//���ú�������Ҫ����block��grid�Ĺ�ϵ
void swap_image(cuda::GpuMat src, cuda::GpuMat dst, int h, int w)
{
	assert(src.cols == w && src.rows == h);
	int uint = 32;
	//�ο�ǰ�����ĵ�block��grid�ļ��㷽����ע�ⲻҪ����GPU����
	dim3 block(uint, uint);
	dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
	printf("grid = %4d %4d %4d\n", grid.x, grid.y, grid.z);
	printf("block= %4d %4d %4d\n", block.x, block.y, block.z);
	swap_image_kernel << <grid, block >> > (src, dst, h, w);
	//ͬ��һ�£���Ϊ���������ܴܺ�
	CHECK_ERROR(hipDeviceSynchronize());
}
int main(int argc, char **argv)
{
	Mat src, dst;
	cuda::GpuMat cu_src, cu_dst;
	int h, w;
	//����argv[1]����ͼƬ���ݣ�BGR��ʽ������
	src = imread(argv[1]);
	//����Ƿ���ȷ����
	if (src.data == NULL)
	{
		cout << "Read image error" << endl;
		return -1;
	}
	h = src.rows; w = src.cols;
	cout << "ͼƬ�ߣ�" << h << ",ͼƬ��" << w << endl;
	//�ϴ�CPUͼ�����ݵ�GPU����hipMalloc��hipMemcpy����Ŷ����ʵupload���������ôд��
	cu_src.upload(src);
	//����GPU�ռ䣬Ҳ���Ե����������룬����������Ҫ���룬Ҫ��Ȼ�ں˺����ᱬ��Ŷ
	cu_dst = cuda::GpuMat(h, w, CV_8UC3, Scalar(0, 0, 0));
	//����CPU�ռ�
	dst = Mat(h, w, CV_8UC3, Scalar(0, 0, 0));
	//���ú���swap_image,�ɸú��������ں˺�����������η����������׳���
	//��Ȼ��Ҳ����ֱ������������ں˺���������̫�����������
	swap_image(cu_src, cu_dst, h, w);
	//����GPU���ݵ�CPU����upload()��Ӧ
	cu_dst.download(dst);
	//��ʾcpuͼ�������װ��openCV������openGL,�ǿ���ֱ����ʾGpuMat
	imshow("dst", dst);
	//�ȴ�����
	waitKey();
	//дͼƬ���ļ�
	if (argc == 3)
		imwrite(argv[2], dst);
	return 0;
}
