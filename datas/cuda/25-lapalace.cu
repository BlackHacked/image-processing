
#include <hip/hip_runtime.h>
extern "C" {

__device__ int KerSobel(int a1, int a2, int a3, int a4, int a5, int a6)
{
    return(a1 + 2 * a2 + a3 - (a4 + 2 * a5 + a6));
}

__global__ void laplacian_filter(unsigned int *lpSrc,unsigned int *lpDst,  int width, int height,int* gc_weight, int amplitude)
{
    int   x = blockIdx.x * blockDim.x + threadIdx.x;
    int  y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx  = width * y + x;
    int xy[9];
    int dr, dg, db;
    int powR, powG, powB;

    if(x >= 1 && x < width - 1 && y >= 1 && y < height - 1)
    {
        /*lpDst[idx] = 8 * lpSrc[idx] -
            lpSrc[idx - 1] - lpSrc[idx + 1] -
            lpSrc[idx - width] - lpSrc[idx + width] -
            lpSrc[idx - width - 1] - lpSrc[idx + width + 1] -
            lpSrc[idx - width + 1] - lpSrc[idx + width - 1];
        */

        xy[0]= lpSrc[idx - width - 1];
        xy[1]= lpSrc[idx - width];
        xy[2]= lpSrc[idx - width + 1];
        xy[3]= lpSrc[idx - 1];
        xy[4]= lpSrc[idx];
        xy[5]= lpSrc[idx + 1];
        xy[6]= lpSrc[idx + width - 1];
        xy[7]= lpSrc[idx + width];
        xy[8]= lpSrc[idx + width + 1];

        dr = dg = db = 0;

        for(int i = 0; i < 9; i ++)
        {
            dr += gc_weight[i] * (0xFF & (xy[i] >> 16));
            dg += gc_weight[i] * (0xFF & (xy[i] >> 8));
            db += gc_weight[i] * (0xFF & (xy[i]));
        }

        /* Calculate power */
        powR = amplitude * dr * dr >> 10; // * amplitude / 1024
        powG = amplitude * dg * dg >> 10;
        powB = amplitude * db * db >> 10;
        if(powR > 255) { powR = 255; }
        if(powG > 255) { powG = 255; }
        if(powB > 255) { powB = 255; }

        lpDst[y * width + x] = (powR << 16) | (powG << 8) | (powB);
     }
}

}