
#include <hip/hip_runtime.h>
extern "C" {

__global__ void rgb2gray(uchar3 *dataIn, unsigned char *dataOut, int imgHeight, int imgWidth)
{
    int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
    int yIndex = threadIdx.y + blockIdx.y * blockDim.y;

    if (xIndex < imgWidth && yIndex < imgHeight)
    {
        uchar3 rgb = dataIn[yIndex * imgWidth + xIndex];

        dataOut[yIndex * imgWidth + xIndex] = 0.299f * rgb.x + 0.587f * rgb.y + 0.114f * rgb.z;
    }
}

}