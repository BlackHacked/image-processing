#include "hip/hip_runtime.h"
extern "C" {


    const int Tile_width = 16;

    __constant__ double filter_d1[81];//Constant memory variable

    texture<unsigned char,2,hipReadModeElementType> texIn; // Input to texture memory

    __global__ void motion_blur(unsigned char* imaged, unsigned char* outputImaged,int width,int height,double* filter){

        int row = blockIdx.y * Tile_width + threadIdx.y;
        int col = blockIdx.x * Tile_width + threadIdx.x;

        if(row < height && col < width){
                //Perform Image convolution 
            double accum = 0;
                for(int fw = 0 ; fw < 9; fw++)
                    for(int fh = 0; fh < 9; fh++)
                    {
                        int ix = ( col - 4 + fw + width)% width;
                        int iy = ( row - 4 + fh + height)%height;
                        accum = accum + (imaged[iy * width + ix] * filter[fw*3 + fh]);
                    }
                accum /= 9;
                unsigned char temp = accum;
                outputImaged[row * width + col] = temp;
        }
    }

    __global__ void motion_blur_1(unsigned char* imaged, unsigned char* outputImaged,int width,int height){

        int row = blockIdx.y * Tile_width + threadIdx.y;
        int col = blockIdx.x * Tile_width + threadIdx.x;

        if(row < height && col < width){
                //Perform Image convolution 
            double accum = 0;
                for(int fw = 0 ; fw < 9; fw++)
                    for(int fh = 0; fh < 9; fh++)
                    {
                        int ix = ( col - 4 + fw + width)% width;
                        int iy = ( row - 4 + fh + height)%height;
                        accum = accum + (imaged[iy * width + ix] * filter_d1[fw*3 + fh]);
                    }
                accum /= 9;
                unsigned char temp = accum;
                outputImaged[row * width + col] = temp;
        }
    }

    __global__ void motion_blur_3(unsigned char* outputImaged,int width,int height,double* filter){

        int row = blockIdx.y * Tile_width + threadIdx.y;
        int col = blockIdx.x * Tile_width + threadIdx.x;

        if(row < height && col < width){
                //Perform Image convolution 
            double accum = 0;
                for(int fw = 0 ; fw < 9; fw++)
                    for(int fh = 0; fh < 9; fh++)
                    {
                        int ix = ( col - 4 + fw + width)% width;
                        int iy = ( row - 4 + fh + height)%height;
                        accum = accum + (tex2D(texIn,ix,iy) * filter[fw*3 + fh]);
                    }
                accum /= 9;
                unsigned char temp = accum;
                outputImaged[row * width + col] = temp;
        }
}

}